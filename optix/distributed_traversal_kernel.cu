#include "hip/hip_runtime.h"
//TODO
#include <optix.h>

#include <stdio.h>

#include "bsdfs/lambertian.hpp"
#include "bsdfs/water.hpp"
#include "moana/core/bsdf_sample_record.hpp"
#include "moana/core/camera.hpp"
#include "moana/core/ray.hpp"
#include "moana/cuda/bsdf.hpp"
#include "moana/driver.hpp"
#include "moana/render/renderer.hpp"
#include "optix_sdk.hpp"
#include "random.hpp"
#include "ray_data.hpp"
#include "util.hpp"

#include "moana/core/coordinates.hpp"

using namespace moana;

// static constexpr float rotationOffset = 0.f / 180.f * M_PI;//旋转环境光贴图

extern "C" {
    __constant__ Renderer::Params params;
}

__forceinline__ __device__ static void setbit(unsigned int &visitedMask, int worldID) {
    visitedMask |= (1 << worldID); 
}

// __device__ static int OPTIX_GetEnterVisibilityIndex(const aabbRecord &aabb, int faceID, const float3 &pointLocal, Ray ray) {
//     float phi, theta, row, column;
//     int directionIndex, coordinateIndex;
//     Vec3 direction;

//     int enterFaceType = faceID / 2;

//     if (enterFaceType == 0 || enterFaceType == 1)
//     {
//         if(ray.direction().x() > 0) {
//             direction = Vec3{ray.direction().y(), ray.direction().z(), ray.direction().x()};
//         } else {
//             direction = Vec3{-ray.direction().y(), -ray.direction().z(), -ray.direction().x()};
//         }

//         column = (pointLocal.y - aabb.m_minY) / (aabb.m_maxY - aabb.m_minY);
//         row = (aabb.m_maxZ - pointLocal.z) / (aabb.m_maxZ - aabb.m_minZ);
//     } 
//     else if (enterFaceType == 2 || enterFaceType == 3) 
//     {
//         if(ray.direction().y() > 0) {
//             direction = Vec3{ray.direction().z(), ray.direction().x(), ray.direction().y()};
//         } else {
//             direction = Vec3{-ray.direction().z(), -ray.direction().x(), -ray.direction().y()};
//         }

//         column = (aabb.m_maxX - pointLocal.x) / (aabb.m_maxX - aabb.m_minX);
//         row = (aabb.m_maxZ - pointLocal.z) / (aabb.m_maxZ - aabb.m_minZ);
//     }
//     else if (enterFaceType == 4 || enterFaceType == 5) 
//     {   
//         if(ray.direction().z() > 0) {
//             direction = Vec3{ray.direction().x(), ray.direction().y(), ray.direction().z()};
//         } else {
//             direction = Vec3{-ray.direction().x(), -ray.direction().y(), -ray.direction().z()};
//         }

//         column = (pointLocal.y - aabb.m_minY) / (aabb.m_maxY - aabb.m_minY);
//         row = (pointLocal.x - aabb.m_minX) / (aabb.m_maxX - aabb.m_minX);
//     }

//     Coordinates::cartesianToSphericalForAABB(direction, &phi, &theta);

//     directionIndex = int(float(aabb.angle) * phi / (2 * M_PI));
//     coordinateIndex = int(row * aabb.height) * aabb.width + int(column * aabb.width);
    
//     return (enterFaceType * (aabb.width * aabb.height * aabb.angle) +  coordinateIndex * aabb.angle + directionIndex);
// }

__device__ static void calculateEnvironmentLighting(WavefrontPathData &path) {
    float4 environmentLight{0.f, 0.f, 0.f, 0.f};

    // Pixels that have already been lit in previous bounces
    float phi, theta;
    Coordinates::cartesianToSpherical(path.direction, &phi, &theta);

    phi += rotationOffset;
    if (phi > 2.f * M_PI) {
        phi -= 2.f * M_PI;
    }

    environmentLight = tex2D<float4>(
        params.envLightTexture,
        phi / (M_PI * 2.f),
        theta / M_PI
    );

    Vec3 environment = Vec3(environmentLight.x, environmentLight.y, environmentLight.z);
    path.throughput = path.throughput * environment;
}

__forceinline__ __device__ static PerRayData *getPRD() {
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<PerRayData *>(util::unpackPointer(u0, u1));
}

extern "C" __global__ void __anyhit__ah()
{   
    if (optixIsTriangleHit()) {
        unsigned int primitiveIndex = optixGetPrimitiveIndex();
        HitGroupData *hitgroupData = reinterpret_cast<HitGroupData *>(optixGetSbtDataPointer());
        
        const int textureIndex = hitgroupData->textureIndex;

        if (textureIndex >= 0)
        {
            int textureIndex0 = hitgroupData->texCoordsIndices[primitiveIndex * 3 + 0];
            int textureIndex1 = hitgroupData->texCoordsIndices[primitiveIndex * 3 + 1];
            int textureIndex2 = hitgroupData->texCoordsIndices[primitiveIndex * 3 + 2];

            float t0x = hitgroupData->texCoords[textureIndex0 * 2 + 0];
            float t0y = hitgroupData->texCoords[textureIndex0 * 2 + 1];

            float t1x = hitgroupData->texCoords[textureIndex1 * 2 + 0];
            float t1y = hitgroupData->texCoords[textureIndex1 * 2 + 1];

            float t2x = hitgroupData->texCoords[textureIndex2 * 2 + 0];
            float t2y = hitgroupData->texCoords[textureIndex2 * 2 + 1];

            const float2 barycentrics = optixGetTriangleBarycentrics();
            const float alpha = barycentrics.x;
            const float beta = barycentrics.y;
            const float gamma = 1.f - alpha - beta;

            float tx = gamma * t0x + alpha * t1x + beta * t2x;
            float ty = gamma * t0y + alpha * t1y + beta * t2y;

            float4 albedo{0.f, 0.f, 0.f, 0.f};
            albedo = tex2D<float4>(
                params.albedoTextures[textureIndex],
                tx,
                ty
            );

            const float opacity = albedo.w;

            // Stochastic alpha test to get an alpha blend effect.
            if (opacity < 0.05f)// No need to calculate an expensive random number if the test is going to fail anyway.
            {
                optixIgnoreIntersection();
            }

        }
    }
}

extern "C" __global__ void __closesthit__ch() {
    PerRayData *prd = getPRD();
    prd->isHit = true;
    prd->t = optixGetRayTmax();

    const float3 rayDirection = optixGetWorldRayDirection();
    const float3 point = getHitPoint();
    prd->point = point;
    prd->pointLocal = optixTransformPointFromWorldToObjectSpace(point);
    prd->originLocal = optixTransformPointFromWorldToObjectSpace(optixGetWorldRayOrigin());
    prd->directionLocal = optixTransformVectorFromWorldToObjectSpace(rayDirection);

    const unsigned int primitiveIndex = optixGetPrimitiveIndex();
    prd->primitiveID = primitiveIndex;
    // prd->instanceID  = optixGetInstanceIndex();
    const Vec3 woWorld = -Vec3(rayDirection.x, rayDirection.y, rayDirection.z);
    Vec3 normal;

   if (optixIsTriangleHit()) {
        HitGroupData *hitgroupData = reinterpret_cast<HitGroupData *>(optixGetSbtDataPointer());
        int normalIndex0 = hitgroupData->normalIndices[primitiveIndex * 3 + 0];

        float n0x = hitgroupData->normals[normalIndex0 * 3 + 0];
        float n0y = hitgroupData->normals[normalIndex0 * 3 + 1];
        float n0z = hitgroupData->normals[normalIndex0 * 3 + 2];

        float3 n0Object{n0x, n0y, n0z};

        float3 n0World = optixTransformNormalFromObjectToWorldSpace(n0Object);

        const Vec3 n0 = normalized(Vec3(n0World.x, n0World.y, n0World.z));

        normal = n0;
    } else {
        const float3 normalFLT3 = normalCubic(primitiveIndex);
        normal = normalized(Vec3(normalFLT3.x, normalFLT3.y, normalFLT3.z));
    }

    if (dot(normal, woWorld) < 0.f) {
        normal = -1.f * normal;
        prd->isInside = true;
    } else {
        prd->isInside = false;
    }

}

extern "C" __global__ void __miss__ms() {

    float3 direction = optixGetWorldRayDirection();
    PerRayData *prd = getPRD();
    prd->isHit = false;
    prd->materialID = -1;
}

extern "C" __global__ void __raygen__rg() {

    const uint3 index = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const int threadIndex = index.y * dim.x + index.x;

    if (threadIndex >= params.pathSize) return;

    WavefrontPathData path;
    path = params.pathDataBuffer[threadIndex];

    if (!path.isValid) { return; }

    const Vec3 origin = path.origin;
    const Vec3 direction = path.direction;

    PerRayData prd;
    
    // intersect with local geometries
    for (int i = 0; i < params.sceneSize; i++) {
        const Renderer::AccelerationStructure &AS = params.accelerationStructures[i];

        if(AS.isProxy) continue;
        if(path.visitedMask >> AS.nodeID & 1) continue; //判断是否已经进行求交

        prd.isHit = false;

        unsigned int p0, p1;
        util::packPointer(&prd, p0, p1);
        optixTrace(
            AS.handle,
            float3{ origin.x(), origin.y(), origin.z() },
            float3{ direction.x(), direction.y(), direction.z() },
            util::Epsilon,
            path.tMax,
            0.f,
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_ENFORCE_ANYHIT,
            0, 1, 0, // SBT params
            p0, p1
        );

        if (prd.isHit) {
            path.tMax = prd.t;
            path.isHit = true;
            path.currentNode = params.worldID;
        } 
    }

    // if (true) {
    //     params.directionBuffer[path.pixelIndex * 3 + 0] = float(path.isHit);
    //     params.directionBuffer[path.pixelIndex * 3 + 1] = float(path.isHit);
    //     params.directionBuffer[path.pixelIndex * 3 + 2] = float(path.isHit);
    // }

    // set visitedMask
    setbit(path.visitedMask, params.worldID);
    
    // intersect with geometry's proxies
    float tMax = path.tMax;
    bool isHit = false;
    bool isInside = false;
    int hitAABBIndex = -1;
    float3 hitPoint;
    for (int i = 0; i < params.sceneSize; i++) {

        const Renderer::AccelerationStructure &AS = params.accelerationStructures[i];

        if(!AS.isProxy) continue; //判断是否为代理加速结构
        if(path.visitedMask >> AS.nodeID & 1) continue; //判断是否已经进行求交

        prd.isHit = false;

        unsigned int p0, p1;
        util::packPointer(&prd, p0, p1);
        optixTrace(
            AS.aabbHandle,
            float3{ origin.x(), origin.y(), origin.z() },
            float3{ direction.x(), direction.y(), direction.z() },
            util::Epsilon,
            tMax,
            0.f,
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_NONE,
            0, 1, 0, // SBT params
            p0, p1
        );

        if (prd.isHit) {
            tMax = prd.t;
            isHit = true;
            path.targetNode = AS.nodeID;
            hitAABBIndex = i;
            isInside = prd.isInside;
            // hitPoint = prd.pointLocal;
        } 
    }

    if (!isHit) path.targetNode = path.currentNode;

    // if(path.isShadowRay) {
        // if(path.isHit) {
        //     path.isValid = false;
        // } 
        // else if (!isHit && !path.isHit) {
        //     const int pixelIndex = path.pixelIndex * 3;
        //     params.directLightingBuffer[pixelIndex + 0] += path.throughput.r();
        //     params.directLightingBuffer[pixelIndex + 1] += path.throughput.g();
        //     params.directLightingBuffer[pixelIndex + 2] += path.throughput.b();
        // }
    // } else {

    if (!isHit && !path.isHit) {
        calculateEnvironmentLighting(path);
        path.isValid = false;
        const int pixelIndex = path.pixelIndex * 3;
        params.envLightingBuffer[pixelIndex + 0] += path.throughput.r();
        params.envLightingBuffer[pixelIndex + 1] += path.throughput.g();
        params.envLightingBuffer[pixelIndex + 2] += path.throughput.b();
    }

    // }

    params.pathDataBuffer[threadIndex] = path;
}
