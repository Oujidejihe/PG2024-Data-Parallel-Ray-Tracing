#include "hip/hip_runtime.h"
#include <optix.h>

#include <stdio.h>
#include <float.h>

#include "bsdfs/lambertian.hpp"
#include "bsdfs/water.hpp"
#include "moana/core/bsdf_sample_record.hpp"
#include "moana/core/camera.hpp"
#include "moana/core/ray.hpp"
#include "moana/cuda/bsdf.hpp"
#include "moana/driver.hpp"
#include "moana/render/renderer.hpp"
#include "optix_sdk.hpp"
#include "random.hpp"
#include "ray_data.hpp"
#include "util.hpp"

#include "sample.hpp"
#include "moana/cuda/triangle.hpp"
#include "moana/core/frame.hpp"

#include "moana/core/coordinates.hpp"

using namespace moana;

// static constexpr float rotationOffset = 0.f / 180.f * M_PI;//旋转环境光贴图

extern "C" {
    __constant__ Renderer::Params params;
}

__forceinline__ __device__ static void setbit(unsigned int &visitedMask, int worldID) {
    visitedMask |= (1 << worldID); 
}

__device__ static void calculateEnvironmentLighting(WavefrontPathData &path) {
    float4 environmentLight{0.f, 0.f, 0.f, 0.f};

    // Pixels that have already been lit in previous bounces
    float phi, theta;
    Coordinates::cartesianToSpherical(path.direction, &phi, &theta);

    phi += rotationOffset;
    if (phi > 2.f * M_PI) {
        phi -= 2.f * M_PI;
    }

    environmentLight = tex2D<float4>(
        params.envLightTexture,
        phi / (M_PI * 2.f),
        theta / M_PI
    );

    Vec3 environment = Vec3(environmentLight.x, environmentLight.y, environmentLight.z);
    path.throughput = path.throughput * environment;
}

__forceinline__ __device__ static PerRayData *getPRD()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<PerRayData *>(util::unpackPointer(u0, u1));
}

extern "C" __global__ void __anyhit__ah()
{   
    if (optixIsTriangleHit()) {
        unsigned int primitiveIndex = optixGetPrimitiveIndex();
        HitGroupData *hitgroupData = reinterpret_cast<HitGroupData *>(optixGetSbtDataPointer());
        
        const int textureIndex = hitgroupData->textureIndex;

        if (textureIndex >= 0)
        {
            int textureIndex0 = hitgroupData->texCoordsIndices[primitiveIndex * 3 + 0];
            int textureIndex1 = hitgroupData->texCoordsIndices[primitiveIndex * 3 + 1];
            int textureIndex2 = hitgroupData->texCoordsIndices[primitiveIndex * 3 + 2];

            float t0x = hitgroupData->texCoords[textureIndex0 * 2 + 0];
            float t0y = hitgroupData->texCoords[textureIndex0 * 2 + 1];

            float t1x = hitgroupData->texCoords[textureIndex1 * 2 + 0];
            float t1y = hitgroupData->texCoords[textureIndex1 * 2 + 1];

            float t2x = hitgroupData->texCoords[textureIndex2 * 2 + 0];
            float t2y = hitgroupData->texCoords[textureIndex2 * 2 + 1];

            const float2 barycentrics = optixGetTriangleBarycentrics();
            const float alpha = barycentrics.x;
            const float beta = barycentrics.y;
            const float gamma = 1.f - alpha - beta;

            float tx = gamma * t0x + alpha * t1x + beta * t2x;
            float ty = gamma * t0y + alpha * t1y + beta * t2y;

            float4 albedo{0.f, 0.f, 0.f, 0.f};
            albedo = tex2D<float4>(
                params.albedoTextures[textureIndex],
                tx,
                ty
            );

            const float opacity = albedo.w;

            // Stochastic alpha test to get an alpha blend effect.
            if (opacity < 0.05f)// No need to calculate an expensive random number if the test is going to fail anyway.
            {
                optixIgnoreIntersection();
            }

        }
    }
}

extern "C" __global__ void __closesthit__ch() {
    PerRayData *prd = getPRD();
    prd->isHit = true;
    prd->t = optixGetRayTmax();

    const float3 rayDirection = optixGetWorldRayDirection();
    const float3 point = getHitPoint();
    prd->point = point;
    prd->pointLocal = optixTransformPointFromWorldToObjectSpace(point);
    prd->originLocal = optixTransformPointFromWorldToObjectSpace(optixGetWorldRayOrigin());
    prd->directionLocal = optixTransformVectorFromWorldToObjectSpace(rayDirection);

    const unsigned int primitiveIndex = optixGetPrimitiveIndex();
    // prd->primitiveID = primitiveIndex;
    prd->instanceID  = optixGetInstanceIndex();
    const Vec3 woWorld = -Vec3(rayDirection.x, rayDirection.y, rayDirection.z);
    Vec3 normal;

   if (optixIsTriangleHit()) {
        HitGroupData *hitgroupData = reinterpret_cast<HitGroupData *>(optixGetSbtDataPointer());
        int normalIndex0 = hitgroupData->normalIndices[primitiveIndex * 3 + 0];

        float n0x = hitgroupData->normals[normalIndex0 * 3 + 0];
        float n0y = hitgroupData->normals[normalIndex0 * 3 + 1];
        float n0z = hitgroupData->normals[normalIndex0 * 3 + 2];

        float3 n0Object{n0x, n0y, n0z};

        float3 n0World = optixTransformNormalFromObjectToWorldSpace(n0Object);

        const Vec3 n0 = normalized(Vec3(n0World.x, n0World.y, n0World.z));

        normal = n0;
    } else {
        const float3 normalFLT3 = normalCubic(primitiveIndex);
        normal = normalized(Vec3(normalFLT3.x, normalFLT3.y, normalFLT3.z));
    }

    if (dot(normal, woWorld) < 0.f) {
        normal = -1.f * normal;
        prd->isInside = true;
    } else {
        prd->isInside = false;
    }

    prd->isInside = !optixIsFrontFaceHit(optixGetHitKind());
}

extern "C" __global__ void __miss__ms()
{
    float3 direction = optixGetWorldRayDirection();
    PerRayData *prd = getPRD();
    prd->isHit = false;
    prd->materialID = -1;
}

extern "C" __global__ void __raygen__rg()
{
    const uint3 index = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const int threadIndex = index.y * dim.x + index.x;

    WavefrontPathData path;
    path = params.pathDataBuffer[threadIndex];

    if(threadIndex >= params.pathSize) return;
    if (!path.isValid) { return; }

    const Vec3 origin = path.origin;
    const Vec3 direction = path.direction;

    PerRayData prd;

    // intersect with local geometries
    for (int i = 0; i < params.sceneSize; i++) {
        setbit(path.visitedMask, params.accelerationStructures[i].nodeID);

        if(params.accelerationStructures[i].isProxy) { continue; }

        prd.isHit = false;

        unsigned int p0, p1;
        util::packPointer(&prd, p0, p1);
        optixTrace(
            params.accelerationStructures[i].handle,
            float3{ origin.x(), origin.y(), origin.z() },
            float3{ direction.x(), direction.y(), direction.z() },
            util::Epsilon,
            path.tMax,
            0.f,
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_ENFORCE_ANYHIT,
            0, 1, 0, // SBT params
            p0, p1
        );

        if (prd.isHit) {
            path.tMax = prd.t;
            path.isHit = true;
            path.currentNode = params.worldID;
        } 
    }
    
    // set visitedMask
    // setbit(path.visitedMask, params.worldID);

    bool isHit = true;
    bool isInside;
    float tMin = 0;
    int count = 0;
    int hitAABBIndex;
    int maxCount = params.maxCount;

    while (isHit && count < maxCount) {

        isHit = false;
        float tMax = path.tMax;

        for (int i = 0; i < params.sceneSize; i++) {
            const Renderer::AccelerationStructure &AS = params.accelerationStructures[i];

            if(!AS.isProxy) continue; //判断是否为代理加速结构
    
            prd.isHit = false;
            
            unsigned int p0, p1;
            util::packPointer(&prd, p0, p1);
            optixTrace(
                AS.aabbHandle,
                float3{ origin.x(), origin.y(), origin.z() },
                float3{ direction.x(), direction.y(), direction.z() },
                tMin + util::Epsilon,
                tMax,
                0.f,
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_NONE,
                0, 1, 0, // SBT params
                p0, p1
            );

            if (prd.isHit) {
                tMax = prd.t;
                isHit = true;
                hitAABBIndex = i;
                isInside = prd.isInside;
            }  
        }

        aabbRecord AABBInfo;
        float aabbX, aabbY, aabbZ;

        if (isHit) {
            AABBInfo = params.accelerationStructures[hitAABBIndex].AABBInfo;
            tMin = tMax;
            aabbX = (AABBInfo.m_maxX - AABBInfo.m_minX);
            aabbY = (AABBInfo.m_maxY - AABBInfo.m_minY);
            aabbZ = (AABBInfo.m_maxZ - AABBInfo.m_minZ);
        }

        if (isHit && !isInside) {
            float phi, theta;
            Vec3 directionLocal = normalized(Vec3(prd.directionLocal.x, prd.directionLocal.y, prd.directionLocal.z));
            Coordinates::cartesianToSphericalForTrain(directionLocal, &phi, &theta);

            const int inputDataOffset = threadIndex * 5 * maxCount + count * 5;

            params.inputDataBuffer[inputDataOffset + 0] = __float2half((prd.pointLocal.x - AABBInfo.m_minX) / aabbX);
            params.inputDataBuffer[inputDataOffset + 1] = __float2half((prd.pointLocal.y - AABBInfo.m_minY) / aabbY);
            params.inputDataBuffer[inputDataOffset + 2] = __float2half((prd.pointLocal.z - AABBInfo.m_minZ) / aabbZ);
            params.inputDataBuffer[inputDataOffset + 3] = __float2half(phi / (2 * M_PI));
            params.inputDataBuffer[inputDataOffset + 4] = __float2half(theta / M_PI);
            
            NNPathData tempPathData;
            //通道1：当前交点的tMax
            tempPathData.throughput[0] = tMax;
            tempPathData.throughput[1] = AABBInfo.m_maxLength;
            tempPathData.throughput[2] = tMax / length(prd.originLocal - prd.pointLocal);

            tempPathData.pixelIndex = path.pixelIndex;
            tempPathData.hitScequnce = count;
            tempPathData.hitAABBID = hitAABBIndex + 1;
            tempPathData.isValid = true;
            // tempPathData.shadowPathID = path.shadowPathID;
            tempPathData.isInside = false;
            tempPathData.instanceID = prd.instanceID;
            tempPathData.pathIndex = params.accelerationStructures[hitAABBIndex].nodeID;

            params.NNPathDataBuffer[threadIndex * maxCount + count] = tempPathData; //TODO: AABBID + 1
            
            count++;
        } 
        else if (isHit && isInside) {

            bool skip = false;
            for(int i = 0; i < count; i++) {
                if (params.NNPathDataBuffer[threadIndex * maxCount + i].hitAABBID == (hitAABBIndex + 1)
                    && params.NNPathDataBuffer[threadIndex * maxCount + i].instanceID == prd.instanceID)
                    skip = true;
            }
            if(skip && count) continue;

            float phi, theta;
            Vec3 directionLocal = Vec3(-prd.directionLocal.x, -prd.directionLocal.y, -prd.directionLocal.z);
            Coordinates::cartesianToSphericalForTrain(normalized(directionLocal), &phi, &theta);

            const int inputDataOffset = threadIndex * 5 * maxCount + count * 5;

            params.inputDataBuffer[inputDataOffset + 0] = __float2half((prd.pointLocal.x - AABBInfo.m_minX) / aabbX);
            params.inputDataBuffer[inputDataOffset + 1] = __float2half((prd.pointLocal.y - AABBInfo.m_minY) / aabbY);
            params.inputDataBuffer[inputDataOffset + 2] = __float2half((prd.pointLocal.z - AABBInfo.m_minZ) / aabbZ);
            params.inputDataBuffer[inputDataOffset + 3] = __float2half(phi / (2 * M_PI));
            params.inputDataBuffer[inputDataOffset + 4] = __float2half(theta / M_PI);
            
            NNPathData tempPathData;
            //通道1：当前交点的tMax
            tempPathData.throughput[0] = tMax;
            tempPathData.throughput[1] = AABBInfo.m_maxLength;
            tempPathData.throughput[2] = tMax / length(prd.originLocal - prd.pointLocal);

            // TODO : 预测的tMax使用有误

            // tempPathData.throughput[2] = path.throughput[2];
            
            tempPathData.pixelIndex = path.pixelIndex;
            tempPathData.hitScequnce = count;
            tempPathData.hitAABBID = hitAABBIndex + 1;
            tempPathData.isValid = true;
            // tempPathData.shadowPathID = path.shadowPathID;
            tempPathData.instanceID = prd.instanceID;
            tempPathData.pathIndex = params.accelerationStructures[hitAABBIndex].nodeID;

            tempPathData.isInside = true;
            tempPathData.normalizedT = tMax / AABBInfo.m_maxLength;

            params.NNPathDataBuffer[threadIndex * maxCount + count] = tempPathData; //TODO: AABBID + 1

            count++;
        } 
        else if (!isHit && count == 0 && !path.isHit) {
            calculateEnvironmentLighting(path);
            path.isValid = false;
            const int pixelIndex = path.pixelIndex * 3;
            params.envLightingBuffer[pixelIndex + 0] += path.throughput.r();
            params.envLightingBuffer[pixelIndex + 1] += path.throughput.g();
            params.envLightingBuffer[pixelIndex + 2] += path.throughput.b();
        }  
    }

    // params.missDirectionBuffer[missDirectionIndex + 0] = direction.x();
    // params.missDirectionBuffer[missDirectionIndex + 1] = direction.y();
    // params.missDirectionBuffer[missDirectionIndex + 2] = direction.z();
    params.pathDataBuffer[threadIndex] = path;
}
